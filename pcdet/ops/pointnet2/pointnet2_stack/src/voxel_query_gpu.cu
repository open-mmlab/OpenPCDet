#include "hip/hip_runtime.h"
/*
Stacked-batch-data version of point grouping, modified from the original implementation of official PointNet++ codes.
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/


#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "group_points_gpu.h"


__global__ void group_points_grad_kernel_stack(int B, int M, int C, int N, int nsample,
    const float *grad_out, const int *idx, const int *idx_batch_cnt, const int *features_batch_cnt, float *grad_features) {
    // :param grad_out: (M1 + M2 ..., C, nsample) tensor of the gradients of the output from forward
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :return:
    //     grad_features: (N1 + N2 ..., C) gradient of the features
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sample_idx = index % nsample;
    int C_idx = (index / nsample) % C;
    int pt_idx = (index / nsample / C);

    if (pt_idx >= M || C_idx >= C || sample_idx >= nsample) return;

    int bs_idx = 0, pt_cnt = idx_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += idx_batch_cnt[k];
        bs_idx = k;
    }

    int features_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) features_batch_start_idx += features_batch_cnt[k];

    grad_out += pt_idx * C * nsample + C_idx * nsample + sample_idx;
    idx += pt_idx * nsample + sample_idx;
    grad_features += (features_batch_start_idx + idx[0]) * C + C_idx;

    atomicAdd(grad_features, grad_out[0]);
}

void group_points_grad_kernel_launcher_stack(int B, int M, int C, int N, int nsample,
    const float *grad_out, const int *idx, const int *idx_batch_cnt, const int *features_batch_cnt, float *grad_features) {
    // :param grad_out: (M1 + M2 ..., C, nsample) tensor of the gradients of the output from forward
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :return:
    //     grad_features: (N1 + N2 ..., C) gradient of the features

    hipError_t err;
    // dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 blocks(DIVUP(M * C * nsample, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel_stack<<<blocks, threads>>>(B, M, C, N, nsample, grad_out, idx, idx_batch_cnt, features_batch_cnt, grad_features);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void group_points_kernel_stack(int B, int M, int C, int nsample,
    const float *features, const int *features_batch_cnt, const int *idx, const int *idx_batch_cnt, float *out) {
    // :param features: (N1 + N2 ..., C) tensor of features to group
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :return:
    //     output: (M1 + M2, C, nsample) tensor
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sample_idx = index % nsample;
    int C_idx = (index / nsample) % C;
    int pt_idx = (index / nsample / C);

    if (pt_idx >= M || C_idx >= C || sample_idx >= nsample) return;

    int bs_idx = 0, pt_cnt = idx_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += idx_batch_cnt[k];
        bs_idx = k;
    }

    int features_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) features_batch_start_idx += features_batch_cnt[k];
    features += features_batch_start_idx * C;

    idx += pt_idx * nsample + sample_idx;
    int in_idx = idx[0] * C + C_idx;
    int out_idx = pt_idx * C * nsample + C_idx * nsample + sample_idx;

    out[out_idx] = features[in_idx];
}


void group_points_kernel_launcher_stack(int B, int M, int C, int nsample,
    const float *features, const int *features_batch_cnt, const int *idx, const int *idx_batch_cnt, float *out) {
    // :param features: (N1 + N2 ..., C) tensor of features to group
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :return:
    //     output: (M1 + M2, C, nsample) tensor

    hipError_t err;
    dim3 blocks(DIVUP(M * C * nsample, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel_stack<<<blocks, threads>>>(B, M, C, nsample, features, features_batch_cnt, idx, idx_batch_cnt, out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
