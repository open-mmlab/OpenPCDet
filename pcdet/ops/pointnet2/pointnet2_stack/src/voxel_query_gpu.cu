#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

#include "voxel_query_gpu.h"
#include "cuda_utils.h"


__global__ void voxel_query_kernel_stack(int M, int R1, int R2, int R3, int nsample, 
            float radius, int z_range, int y_range, int x_range, const float *new_xyz, 
            const float *xyz, const int *new_coords, const int *point_indices, int *idx) {
    // :param new_coords: (M1 + M2 ..., 4) centers of the ball query
    // :param point_indices: (B, Z, Y, X)
    // output:
    //      idx: (M1 + M2, nsample)
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= M) return;
    
    new_xyz += pt_idx * 3;
    new_coords += pt_idx * 4;
    idx += pt_idx * nsample;

    hiprandState state;
    hiprand_init(pt_idx, 0, 0, &state);
    
    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int batch_idx = new_coords[0];
    int new_coords_z = new_coords[1];
    int new_coords_y = new_coords[2];
    int new_coords_x = new_coords[3];
    
    int cnt = 0;
    int cnt2 = 0;
    // for (int dz = -1*z_range; dz <= z_range; ++dz) {
    for (int dz = -1*z_range; dz <= z_range; ++dz) {
        int z_coord = new_coords_z + dz;
        if (z_coord < 0 || z_coord >= R1) continue;

        for (int dy = -1*y_range; dy <= y_range; ++dy) {
            int y_coord = new_coords_y + dy;
            if (y_coord < 0 || y_coord >= R2) continue;

            for (int dx = -1*x_range; dx <= x_range; ++dx) {
                int x_coord = new_coords_x + dx;
                if (x_coord < 0 || x_coord >= R3) continue;

                int index = batch_idx * R1 * R2 * R3 + \
                            z_coord * R2 * R3 + \
                            y_coord * R3 + \
                            x_coord;
                int neighbor_idx = point_indices[index];
                if (neighbor_idx < 0) continue;
                
                float x_per = xyz[neighbor_idx*3 + 0];
                float y_per = xyz[neighbor_idx*3 + 1];
                float z_per = xyz[neighbor_idx*3 + 2];

                float dist2 = (x_per - new_x) * (x_per - new_x) + (y_per - new_y) * (y_per - new_y) + (z_per - new_z) * (z_per - new_z);

                if (dist2 > radius2) continue;
                
                ++cnt2;

                if (cnt < nsample) {
                    if (cnt == 0) {
                        for (int l = 0; l < nsample; ++l) {
                            idx[l] = neighbor_idx;
                        }
                    }
                    idx[cnt] = neighbor_idx;
                    ++cnt;
                }
                // else {
                //     float rnd = hiprand_uniform(&state);
                //     if (rnd < (float(nsample) / cnt2)) {
                //         int insertidx = ceilf(hiprand_uniform(&state) * nsample) - 1;
                //         idx[insertidx] = neighbor_idx;
                //     }
                // }
            }
        }
    }
   if (cnt == 0) idx[0] = -1;
}


void voxel_query_kernel_launcher_stack(int M, int R1, int R2, int R3, int nsample,
    float radius, int z_range, int y_range, int x_range, const float *new_xyz, 
    const float *xyz, const int *new_coords, const int *point_indices, int *idx) {
    // :param new_coords: (M1 + M2 ..., 4) centers of the voxel query
    // :param point_indices: (B, Z, Y, X) 
    // output:
    //      idx: (M1 + M2, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    voxel_query_kernel_stack<<<blocks, threads>>>(M, R1, R2, R3, nsample, radius, z_range, y_range, x_range, new_xyz, xyz, new_coords, point_indices, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
