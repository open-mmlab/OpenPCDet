/*
RoI-aware point cloud feature pooling
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/



#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
// #define DEBUG


__device__ inline void lidar_to_local_coords(float shift_x, float shift_y, float rot_angle, float &local_x, float &local_y){
    float cosa = cos(-rot_angle), sina = sin(-rot_angle);
    local_x = shift_x * cosa + shift_y * (-sina);
    local_y = shift_x * sina + shift_y * cosa;
}


__device__ inline int check_pt_in_box3d(const float *pt, const float *box3d, float &local_x, float &local_y){
    // param pt: (x, y, z)
    // param box3d: [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center

    const float MARGIN = 1e-5;
    float x = pt[0], y = pt[1], z = pt[2];
    float cx = box3d[0], cy = box3d[1], cz = box3d[2];
    float dx = box3d[3], dy = box3d[4], dz = box3d[5], rz = box3d[6];

    if (fabsf(z - cz) > dz / 2.0) return 0;
    lidar_to_local_coords(x - cx, y - cy, rz, local_x, local_y);
    float in_flag = (fabs(local_x) < dx / 2.0 + MARGIN) & (fabs(local_y) < dy / 2.0 + MARGIN);
    return in_flag;
}

__device__ inline int check_pt_in_box_bev(const float *pt, const float *box3d, float &local_x, float &local_y){
    // param pt: (x, y)
    // param box3d: [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center

    const float MARGIN = 1e-5;
    float x = pt[0], y = pt[1];
    float cx = box3d[0], cy = box3d[1];
    float dx = box3d[3], dy = box3d[4], rz = box3d[6];

    lidar_to_local_coords(x - cx, y - cy, rz, local_x, local_y);
    float in_flag = (fabs(local_x) < dx / 2.0 + MARGIN) & (fabs(local_y) < dy / 2.0 + MARGIN);
    return in_flag;
}

__global__ void generate_pts_mask_for_box3d(int boxes_num, int pts_num, int out_x, int out_y, int out_z,
    const float *rois, const float *pts, int *pts_mask){
    // params rois: [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center
    // params pts: (npoints, 3) [x, y, z]
    // params pts_mask: (N, npoints): -1 means point doesnot in this box, otherwise: encode (x_idxs, y_idxs, z_idxs) by binary bit
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    if (pt_idx >= pts_num || box_idx >= boxes_num) return;

    pts += pt_idx * 3;
    rois += box_idx * 7;
    pts_mask += box_idx * pts_num + pt_idx;

    float local_x = 0, local_y = 0;
    int cur_in_flag = check_pt_in_box3d(pts, rois, local_x, local_y);

    pts_mask[0] = -1;
    if (cur_in_flag > 0){
        float local_z = pts[2] - rois[2];
        float dx = rois[3], dy = rois[4], dz = rois[5];

        float x_res = dx / out_x;
        float y_res = dy / out_y;
        float z_res = dz / out_z;

        unsigned int x_idx = int((local_x + dx / 2) / x_res);
        unsigned int y_idx = int((local_y + dy / 2) / y_res);
        unsigned int z_idx = int((local_z + dz / 2) / z_res);

        x_idx = min(max(x_idx, 0), out_x - 1);
        y_idx = min(max(y_idx, 0), out_y - 1);
        z_idx = min(max(z_idx, 0), out_z - 1);

        unsigned int idx_encoding = (x_idx << 16) + (y_idx << 8) + z_idx;
        pts_mask[0] = idx_encoding;
    }
}


__global__ void collect_inside_pts_for_box3d(int boxes_num, int pts_num, int max_pts_each_voxel,
    int out_x, int out_y, int out_z, const int *pts_mask, int *pts_idx_of_voxels){
    // params pts_mask: (N, npoints)  0 or 1
    // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)

    int box_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (box_idx >= boxes_num) return;

    int max_num_pts = max_pts_each_voxel - 1;  // index 0 is the counter
    pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel;

    for (int k = 0; k < pts_num; k++){
        if (pts_mask[box_idx * pts_num + k] != -1){
            unsigned int idx_encoding = pts_mask[box_idx * pts_num + k];
            unsigned int x_idx = (idx_encoding >> 16) & 0xFF;
            unsigned int y_idx = (idx_encoding >> 8) & 0xFF;
            unsigned int z_idx = idx_encoding & 0xFF;
            unsigned int base_offset = x_idx * out_y * out_z * max_pts_each_voxel + y_idx * out_z * max_pts_each_voxel + z_idx * max_pts_each_voxel;
            unsigned int cnt = pts_idx_of_voxels[base_offset];
            if (cnt < max_num_pts){
                pts_idx_of_voxels[base_offset + cnt + 1] = k;
                pts_idx_of_voxels[base_offset]++;
            }
#ifdef DEBUG
        printf("collect: pts_%d, idx(%d, %d, %d), idx_encoding=%x\n",
            k, x_idx, y_idx, z_idx, idx_encoding);
#endif

        }
    }
}


__global__ void roiaware_maxpool3d(int boxes_num, int pts_num, int channels, int max_pts_each_voxel, int out_x,
    int out_y, int out_z, const float *pts_feature, const int *pts_idx_of_voxels, float *pooled_features, int *argmax){
    // params pts_feature: (npoints, C)
    // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel), index 0 is the counter
    // params pooled_features: (N, out_x, out_y, out_z, C)
    // params argmax: (N, out_x, out_y, out_z, C)

    int box_idx = blockIdx.z;
    int channel_idx = blockIdx.y;
    int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

    int x_idx = voxel_idx_flat / (out_y * out_z);
    int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
    int z_idx = voxel_idx_flat % out_z;
    if (box_idx >= boxes_num || channel_idx >= channels|| x_idx >= out_x || y_idx >= out_y || z_idx >= out_z) return;

#ifdef DEBUG
    printf("src pts_idx_of_voxels: (%p, ), argmax: %p\n", pts_idx_of_voxels, argmax);
#endif

    int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
    pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel + offset_base * max_pts_each_voxel;
    pooled_features += box_idx * out_x * out_y * out_z * channels + offset_base * channels + channel_idx;
    argmax += box_idx * out_x * out_y * out_z * channels + offset_base * channels + channel_idx;

    int argmax_idx = -1;
    float max_val = -1e50;

    int total_pts = pts_idx_of_voxels[0];

    for (int k = 1; k <= total_pts; k++){
        if (pts_feature[pts_idx_of_voxels[k] * channels + channel_idx] > max_val){
            max_val = pts_feature[pts_idx_of_voxels[k] * channels + channel_idx];
            argmax_idx = pts_idx_of_voxels[k];
        }
    }

    if (argmax_idx != -1){
        pooled_features[0] = max_val;
    }
    argmax[0] = argmax_idx;

#ifdef DEBUG
    printf("channel_%d idx(%d, %d, %d), argmax_idx=(%d, %.3f), total=%d, after pts_idx: %p, argmax: (%p, %d)\n",
        channel_idx, x_idx, y_idx, z_idx, argmax_idx, max_val, total_pts, pts_idx_of_voxels, argmax, argmax_idx);
#endif
}


__global__ void roiaware_avgpool3d(int boxes_num, int pts_num, int channels, int max_pts_each_voxel, int out_x,
    int out_y, int out_z, const float *pts_feature, const int *pts_idx_of_voxels, float *pooled_features){
    // params pts_feature: (npoints, C)
    // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel), index 0 is the counter
    // params pooled_features: (N, out_x, out_y, out_z, C)
    // params argmax: (N, out_x, out_y, out_z, C)

    int box_idx = blockIdx.z;
    int channel_idx = blockIdx.y;
    int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

    int x_idx = voxel_idx_flat / (out_y * out_z);
    int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
    int z_idx = voxel_idx_flat % out_z;
    if (box_idx >= boxes_num || channel_idx >= channels|| x_idx >= out_x || y_idx >= out_y || z_idx >= out_z) return;

    int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
    pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel + offset_base * max_pts_each_voxel;
    pooled_features += box_idx * out_x * out_y * out_z * channels + offset_base * channels + channel_idx;

    float sum_val = 0;
    int total_pts = pts_idx_of_voxels[0];

    for (int k = 1; k <= total_pts; k++){
        sum_val += pts_feature[pts_idx_of_voxels[k] * channels + channel_idx];
    }

    if (total_pts > 0){
        pooled_features[0] = sum_val / total_pts;
    }
}


void roiaware_pool3d_launcher(int boxes_num, int pts_num, int channels, int max_pts_each_voxel, int out_x, int out_y, int out_z,
    const float *rois, const float *pts, const float *pts_feature, int *argmax, int *pts_idx_of_voxels, float *pooled_features, int pool_method){
    // params rois: (N, 7) [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center
    // params pts: (npoints, 3) [x, y, z]
    // params pts_feature: (npoints, C)
    // params argmax: (N, out_x, out_y, out_z, C)
    // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)
    // params pooled_features: (N, out_x, out_y, out_z, C)
    // params pool_method: 0: max_pool 1: avg_pool

    int *pts_mask = NULL;
    hipMalloc(&pts_mask, boxes_num * pts_num * sizeof(int));  // (N, M)
    hipMemset(pts_mask, -1, boxes_num * pts_num * sizeof(int));

    dim3 blocks_mask(DIVUP(pts_num, THREADS_PER_BLOCK), boxes_num);
    dim3 threads(THREADS_PER_BLOCK);
    generate_pts_mask_for_box3d<<<blocks_mask, threads>>>(boxes_num, pts_num, out_x, out_y, out_z, rois, pts, pts_mask);

    // TODO: Merge the collect and pool functions, SS

    dim3 blocks_collect(DIVUP(boxes_num, THREADS_PER_BLOCK));
    collect_inside_pts_for_box3d<<<blocks_collect, threads>>>(boxes_num, pts_num, max_pts_each_voxel,
        out_x, out_y, out_z, pts_mask, pts_idx_of_voxels);

    dim3 blocks_pool(DIVUP(out_x * out_y * out_z, THREADS_PER_BLOCK), channels, boxes_num);
    if (pool_method == 0){
        roiaware_maxpool3d<<<blocks_pool, threads>>>(boxes_num, pts_num, channels, max_pts_each_voxel, out_x, out_y, out_z,
            pts_feature, pts_idx_of_voxels, pooled_features, argmax);
    }
    else if (pool_method == 1){
        roiaware_avgpool3d<<<blocks_pool, threads>>>(boxes_num, pts_num, channels, max_pts_each_voxel, out_x, out_y, out_z,
            pts_feature, pts_idx_of_voxels, pooled_features);
    }


    hipFree(pts_mask);

#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}


__global__ void roiaware_maxpool3d_backward(int boxes_num, int channels, int out_x, int out_y, int out_z,
    const int *argmax, const float *grad_out, float *grad_in){
    // params argmax: (N, out_x, out_y, out_z, C)
    // params grad_out: (N, out_x, out_y, out_z, C)
    // params grad_in: (npoints, C), return value

    int box_idx = blockIdx.z;
    int channel_idx = blockIdx.y;
    int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

    int x_idx = voxel_idx_flat / (out_y * out_z);
    int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
    int z_idx = voxel_idx_flat % out_z;
    if (box_idx >= boxes_num || channel_idx >= channels|| x_idx >= out_x || y_idx >= out_y || z_idx >= out_z) return;

    int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
    argmax += box_idx * out_x * out_y * out_z * channels + offset_base * channels + channel_idx;
    grad_out += box_idx * out_x * out_y * out_z * channels + offset_base * channels + channel_idx;

    if (argmax[0] == -1) return;

    atomicAdd(grad_in + argmax[0] * channels + channel_idx, grad_out[0] * 1);
}


__global__ void roiaware_avgpool3d_backward(int boxes_num, int channels, int out_x, int out_y, int out_z,
    int max_pts_each_voxel, const int *pts_idx_of_voxels, const float *grad_out, float *grad_in){
    // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)
    // params grad_out: (N, out_x, out_y, out_z, C)
    // params grad_in: (npoints, C), return value

    int box_idx = blockIdx.z;
    int channel_idx = blockIdx.y;
    int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

    int x_idx = voxel_idx_flat / (out_y * out_z);
    int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
    int z_idx = voxel_idx_flat % out_z;
    if (box_idx >= boxes_num || channel_idx >= channels|| x_idx >= out_x || y_idx >= out_y || z_idx >= out_z) return;

    int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
    pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel + offset_base * max_pts_each_voxel;
    grad_out += box_idx * out_x * out_y * out_z * channels + offset_base * channels + channel_idx;


    int total_pts = pts_idx_of_voxels[0];
    float cur_grad = 1 / fmaxf(float(total_pts), 1.0);
    for (int k = 1; k <= total_pts; k++){
        atomicAdd(grad_in + pts_idx_of_voxels[k] * channels + channel_idx, grad_out[0] * cur_grad);
    }
}


void roiaware_pool3d_backward_launcher(int boxes_num, int out_x, int out_y, int out_z, int channels, int max_pts_each_voxel,
    const int *pts_idx_of_voxels, const int *argmax, const float *grad_out, float *grad_in, int pool_method){
    // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)
    // params argmax: (N, out_x, out_y, out_z, C)
    // params grad_out: (N, out_x, out_y, out_z, C)
    // params grad_in: (npoints, C), return value
    // params pool_method: 0: max_pool, 1: avg_pool

    dim3 blocks(DIVUP(out_x * out_y * out_z, THREADS_PER_BLOCK), channels, boxes_num);
    dim3 threads(THREADS_PER_BLOCK);
    if (pool_method == 0){
        roiaware_maxpool3d_backward<<<blocks, threads>>>(
            boxes_num, channels, out_x, out_y, out_z, argmax, grad_out, grad_in
        );
    }
    else if (pool_method == 1){
        roiaware_avgpool3d_backward<<<blocks, threads>>>(
            boxes_num, channels, out_x, out_y, out_z, max_pts_each_voxel, pts_idx_of_voxels, grad_out, grad_in
        );
    }

}


__global__ void points_in_boxes_kernel(int batch_size, int boxes_num, int pts_num, const float *boxes,
    const float *pts, int *box_idx_of_points){
    // params boxes: (B, N, 7) [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center
    // params pts: (B, npoints, 3) [x, y, z] in LiDAR coordinate
    // params boxes_idx_of_points: (B, npoints), default -1

    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= batch_size || pt_idx >= pts_num) return;

    boxes += bs_idx * boxes_num * 7;
    pts += bs_idx * pts_num * 3 + pt_idx * 3;
    box_idx_of_points += bs_idx * pts_num + pt_idx;

    float local_x = 0, local_y = 0;
    int cur_in_flag = 0;
    for (int k = 0; k < boxes_num; k++){
        cur_in_flag = check_pt_in_box3d(pts, boxes + k * 7, local_x, local_y);
        if (cur_in_flag){
            box_idx_of_points[0] = k;
            break;
        }
    }
}


void points_in_boxes_launcher(int batch_size, int boxes_num, int pts_num, const float *boxes,
    const float *pts, int *box_idx_of_points){
    // params boxes: (B, N, 7) [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center
    // params pts: (B, npoints, 3) [x, y, z]
    // params boxes_idx_of_points: (B, npoints), default -1
    hipError_t err;

    dim3 blocks(DIVUP(pts_num, THREADS_PER_BLOCK), batch_size);
    dim3 threads(THREADS_PER_BLOCK);
    points_in_boxes_kernel<<<blocks, threads>>>(batch_size, boxes_num, pts_num, boxes, pts, box_idx_of_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}


__global__ void points_in_boxes_bev_kernel(int batch_size, int boxes_num, int pts_num, const float *boxes,
    const float *pts, int *box_idx_of_points){
    // params boxes: (B, N, 7) [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center
    // params pts: (B, npoints, 2) [x, y] in LiDAR coordinate
    // params boxes_idx_of_points: (B, npoints), default -1

    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= batch_size || pt_idx >= pts_num) return;

    boxes += bs_idx * boxes_num * 7;
    pts += bs_idx * pts_num * 2 + pt_idx * 2;
    box_idx_of_points += bs_idx * pts_num + pt_idx;

    float local_x = 0, local_y = 0;
    int cur_in_flag = 0;
    for (int k = 0; k < boxes_num; k++){
        cur_in_flag = check_pt_in_box_bev(pts, boxes + k * 7, local_x, local_y);
        if (cur_in_flag){
            box_idx_of_points[0] = k;
            break;
        }
    }
}


void points_in_boxes_bev_launcher(int batch_size, int boxes_num, int pts_num, const float *boxes,
    const float *pts, int *box_idx_of_points){
    // params boxes: (B, N, 7) [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center
    // params pts: (B, npoints, 3) [x, y, z]
    // params boxes_idx_of_points: (B, npoints), default -1
    hipError_t err;

    dim3 blocks(DIVUP(pts_num, THREADS_PER_BLOCK), batch_size);
    dim3 threads(THREADS_PER_BLOCK);
    points_in_boxes_bev_kernel<<<blocks, threads>>>(batch_size, boxes_num, pts_num, boxes, pts, box_idx_of_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}

__global__ void bev_in_boxes_kernel(float x_min, float x_max, float y_min, float y_max,
                                     int boxes_num, int batch_size, int x_inds_length, int y_inds_length,
                                     const float *boxes, const float *bev_coords, int *bev_indices){
    /*
    Args:
        boxes: [B, N, 7]
        bev_coords: [X, Y, 2]
        bev_indices: [B, X, Y]
    */

    int bs_idx = blockIdx.x;
    int box_idx = threadIdx.x;
    if (bs_idx >= batch_size || box_idx >= boxes_num) return;

    boxes += bs_idx * boxes_num * 7;
    bev_indices += bs_idx * x_inds_length * y_inds_length;

    float cx = boxes[box_idx * 7 + 0];
    float cy = boxes[box_idx * 7 + 1];
    float dx = boxes[box_idx * 7 + 3];
    float dy = boxes[box_idx * 7 + 4];
    float rz = boxes[box_idx * 7 + 6];
    if (dx == 0 || dy == 0) return;

    const float MARGIN = 0.001;
    float x_length = x_max - x_min;
    float y_length = y_max - y_min;

    float r = dx * 0.5 + dy * 0.5;
    float cosa = cos(-rz), sina = sin(-rz);

    int search_x_min = floor(x_inds_length * ((cx - r - x_min) / x_length));
    if (search_x_min < 0) search_x_min = 0;
    if (search_x_min >= x_inds_length) search_x_min = x_inds_length - 1;
    int search_x_max = ceil(x_inds_length * ((cx + r - x_min) / x_length));
    if (search_x_max < 0) search_x_max = 0;
    if (search_x_max >= x_inds_length) search_x_max = x_inds_length - 1;

    int search_y_min = floor(y_inds_length * ((cy - r - y_min) / y_length));
    if (search_y_min < 0) search_y_min = 0;
    if (search_y_min >= y_inds_length) search_y_min = y_inds_length - 1;
    int search_y_max = ceil(y_inds_length * ((cy + r - y_min) / y_length));
    if (search_y_max < 0) search_y_max = 0;
    if (search_y_max >= y_inds_length) search_y_max = y_inds_length - 1;

    for (int xi = search_x_min; xi <= search_x_max; ++xi){
        for (int yi = search_y_min; yi <= search_y_max; ++yi){
            float x_coords = bev_coords[xi * y_inds_length * 2 + yi * 2 + 0];
            float y_coords = bev_coords[xi * y_inds_length * 2 + yi * 2 + 1];
            float local_x = (x_coords - cx) * cosa + (y_coords - cy) * (-sina);
            float local_y = (x_coords - cx) * sina + (y_coords - cy) * cosa;
            float in_flag = (fabs(local_x) < dx / 2.0 + MARGIN) & (fabs(local_y) < dy / 2.0 + MARGIN);
            if (in_flag) bev_indices[xi * y_inds_length + yi] = box_idx;
        }
    }

}

void bev_in_boxes_launcher(float x_min, float x_max, float y_min, float y_max,
                            int boxes_num, int batch_size, int x_inds_length, int y_inds_length,
                            const float *boxes, const float *bev_coords, int *bev_indices){
    hipError_t err;

    dim3 blocks(batch_size);
    dim3 threads(THREADS_PER_BLOCK);
    bev_in_boxes_kernel<<<blocks, threads>>>(x_min, x_max, y_min, y_max,
        boxes_num, batch_size, x_inds_length, y_inds_length,
        boxes, bev_coords, bev_indices);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}